#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_sparse_mmcsr<float>(const int M, const int N, const int K,
    const float alpha, const float* A,
    const int nnz, const float* B_nonzero_buf, const int* B_idx_pointer_buf, const int* B_nonzero_idx_buf,
    const float beta,float* C){
	// For cuSPARSE, dense matrix A & C is column major, so we can instead compute C'=B'*A'
	// A & C are intrinsically transposed since c/c++ array is row major

	// as sparse B is expressed in CSR format, and CSC of B'is the same with CSR of B
	// use cusparse<t>cscmm if available, if not, transpose B (transposing may not affect the performance as above reason)
	//NON-BLOCKING NON-BLOCKING NON-BLOCKING
	CUSPARSE_CHECK(hipsparseScsrmm(Caffe::cusparse_handle(),HIPSPARSE_OPERATION_TRANSPOSE,
			//N,M,K,nnz, &alpha,
			K,M,N,nnz, &alpha,//dimensions K,M,N refer to the ones before op(A)
			Caffe::cusparse_matdescr(), B_nonzero_buf, B_idx_pointer_buf, B_nonzero_idx_buf,
			A,K,&beta,C,N
			));
}

template <>
void caffe_gpu_sparse_mmcsr<double>(const int M, const int N, const int K,
    const double alpha, const double* A,
    const int nnz, const double* B_nonzero_buf, const int* B_idx_pointer_buf, const int* B_nonzero_idx_buf,
    const double beta,double* C){
	//NON-BLOCKING NON-BLOCKING NON-BLOCKING
	CUSPARSE_CHECK(hipsparseDcsrmm(Caffe::cusparse_handle(),HIPSPARSE_OPERATION_TRANSPOSE,
				N,M,K,nnz, &alpha,
				Caffe::cusparse_matdescr(), B_nonzero_buf, B_idx_pointer_buf, B_nonzero_idx_buf,
				A,K,&beta,C,N
				));
}

template <>
void caffe_gpu_sparse_csrmm<float>(const int M, const int N, const int K,
    const float alpha,
    const int nnz, const float* A_nonzero_buf, const int* A_idx_pointer_buf, const int* A_nonzero_idx_buf,
    const float* B,
    const float beta,
    float* C,float *transpose_C){
	CUSPARSE_CHECK(hipsparseScsrmm2(Caffe::cusparse_handle(),HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_TRANSPOSE,
				M,N,K,nnz, &alpha,
				Caffe::cusparse_matdescr(), A_nonzero_buf, A_idx_pointer_buf, A_nonzero_idx_buf,
				B,N,&beta,transpose_C,M
				));
	//transpose C
	const float one = 1;
	const float zero = 0;
	CUBLAS_CHECK(hipblasSgeam(Caffe::cublas_handle(),HIPBLAS_OP_T, HIPBLAS_OP_T,
			N,M,&one,transpose_C,M,&zero,transpose_C,M,C,N));
}
template <>
void caffe_gpu_sparse_csrmm<double>(const int M, const int N, const int K,
    const double alpha,
    const int nnz, const double* A_nonzero_buf, const int* A_idx_pointer_buf, const int* A_nonzero_idx_buf,
    const double* B,
    const double beta,
    double* C,double *transpose_C){
	CUSPARSE_CHECK(hipsparseDcsrmm2(Caffe::cusparse_handle(),HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_TRANSPOSE,
				M,N,K,nnz, &alpha,
				Caffe::cusparse_matdescr(), A_nonzero_buf, A_idx_pointer_buf, A_nonzero_idx_buf,
				B,N,&beta,transpose_C,M
				));
	//transpose C
	const double one = 1;
	const double zero = 0;
	CUBLAS_CHECK(hipblasDgeam(Caffe::cublas_handle(),HIPBLAS_OP_T, HIPBLAS_OP_T,
			N,M,&one,transpose_C,M,&zero,transpose_C,M,C,N));
}

template <>
void caffe_gpu_sparse_dense2csr<float>(const int M, const int N,
    const float* A, int* nnzPerRow,
    float* A_nonzero_buf, int* A_idx_pointer_buf, int* A_nonzero_idx_buf, int *nnz_total){
	//cusparse<t>nnz() NON-BLOCKING
	//int nnz_total = 0;
	CUSPARSE_CHECK(hipsparseSnnz(Caffe::cusparse_handle(),
			HIPSPARSE_DIRECTION_COLUMN,
			N,M,
			Caffe::cusparse_matdescr(),
			A,N,
			nnzPerRow,//per row for c style row-major matrix
			nnz_total
			));

	CUSPARSE_CHECK(hipsparseSdense2csc(Caffe::cusparse_handle(),
			N,M,
			Caffe::cusparse_matdescr(),
			A,N,
			nnzPerRow,//per row for c style row-major matrix
			A_nonzero_buf,A_nonzero_idx_buf,A_idx_pointer_buf
			));
}

template <>
void caffe_gpu_sparse_dense2csr<double>(const int M, const int N,
    const double* A, int* nnzPerRow,
    double* A_nonzero_buf, int* A_idx_pointer_buf, int* A_nonzero_idx_buf,int *nnz_total){
	//cusparse<t>nnz() NON-BLOCKING
	//int nnz_total = 0;
	CUSPARSE_CHECK(hipsparseDnnz(Caffe::cusparse_handle(),
			HIPSPARSE_DIRECTION_COLUMN,
			N,M,
			Caffe::cusparse_matdescr(),
			A,N,
			nnzPerRow,//per row for c style row-major matrix
			nnz_total
			));

	CUSPARSE_CHECK(hipsparseDdense2csc(Caffe::cusparse_handle(),
			N,M,
			Caffe::cusparse_matdescr(),
			A,N,
			nnzPerRow,//per row for c style row-major matrix
			A_nonzero_buf,A_nonzero_idx_buf,A_idx_pointer_buf
			));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}




template  <typename Dtype>
__global__ void zerout_kernel(void * mutable_gpu_data, int count, Dtype thre){
	//Dtype thre = Dtype(th);
	Dtype* data_ptr_tmp =  static_cast<Dtype*>(mutable_gpu_data);
		//  for(int i=0;i<count;i++){
		//	  if(data_ptr_tmp[i]<thre && data_ptr_tmp[i]>(-thre)){
		//		  data_ptr_tmp[i]=0;
		//	  }
		//  }
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	while(tid<count){
		if(data_ptr_tmp[tid]<thre && data_ptr_tmp[tid]>(-thre)){
			data_ptr_tmp[tid] = 0;
		}
		tid += gridDim.x*blockDim.x;
	}
}

template <typename Dtype>
void caffe_gpu_zerout(void * mutable_gpu_data, const int count, Dtype th){
	zerout_kernel<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(mutable_gpu_data,  count,  th);
}

template void caffe_gpu_zerout<int>(void * mutable_gpu_data, const int count, int th);
template void caffe_gpu_zerout<unsigned int>(void * mutable_gpu_data, const int count, unsigned int th);
template void caffe_gpu_zerout<float>(void * mutable_gpu_data, const int count, float th);
template void caffe_gpu_zerout<double>(void * mutable_gpu_data, const int count, double th);

//template <>
//void caffe_gpu_zerout<int>(void * mutable_gpu_data, int count, int th){
//	zerout_kernel<<<32768,256>>>(mutable_gpu_data,  count,  th);
//}
//
//template <>
//void caffe_gpu_zerout<float>(void * mutable_gpu_data, int count, float th){
//	zerout_kernel<<<32768,256>>>(mutable_gpu_data,  count,  th);
//}
//
//template <>
//void caffe_gpu_zerout<double>(void * mutable_gpu_data, int count, double th){
//	zerout_kernel<<<32768,256>>>(mutable_gpu_data,  count,  th);
//}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y, int stride) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, stride, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y, int stride) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, stride, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

//Usage: dim3 block(c,1); dim3 thread(1,n); col_group_lasso_kernel<<<block,thread>>>(n,c,x,y);
template  <typename Dtype>
__global__ void col_group_lasso_kernel(const int n, const int c, const Dtype *x, Dtype* y){
	int n_offset = 0;
		//initialize y
		while(n_offset<n){
			//int idx1 = (n_offset+threadIdx.y)*gridDim.x+blockIdx.x;
			int idx1 = (n_offset+threadIdx.y)*c+blockIdx.x;
			if(n_offset+threadIdx.y < n){//BUG: THE N MUST BE MULTIPLE TIMES OF BLOCKDIM.Y IN CURRENT IMPLEMENTATION !!!
				y[idx1] = x[idx1]*x[idx1];
			}
			n_offset += blockDim.y;
		}
		__syncthreads();

		//sum along columns
		n_offset=0;
		Dtype res = 0;
		while(n_offset<n){
			int len = (n_offset + blockDim.y)<n ? blockDim.y : (n-n_offset);//valid threads to process
			while(len/2>0){
				if(threadIdx.y<len/2){
					//int idx1 = (n_offset+threadIdx.y)*gridDim.x+blockIdx.x;
					//int idx2 = (n_offset+threadIdx.y+(len+1)/2)*gridDim.x+blockIdx.x;
					int idx1 = (n_offset+threadIdx.y)*c+blockIdx.x;
					int idx2 = (n_offset+threadIdx.y+(len+1)/2)*c+blockIdx.x;
					y[idx1] += y[idx2];
				}
				__syncthreads();
				len=(len+1)/2;
			}

			//res += y[n_offset*gridDim.x+blockIdx.x];
			res += y[n_offset*c+blockIdx.x];
			n_offset += blockDim.y;
		}
		__syncthreads();

		//copy
		n_offset=0;
		while(n_offset<n){
			//int idx1 = (n_offset+threadIdx.y)*gridDim.x+blockIdx.x;
			int idx1 = (n_offset+threadIdx.y)*c + blockIdx.x;
			if(n_offset+threadIdx.y < n){
				if(res){
					y[idx1] = Dtype(sqrt(res));
				}else{
					y[idx1] = Dtype(0);
				}
			}
		  	n_offset += blockDim.y;
		}
}

//Usage: dim3 block(1,n); dim3 thread(c,1); row_group_lasso_kernel<<<block,thread>>>(n,c,x,y);
template  <typename Dtype>
__global__ void row_group_lasso_kernel(const int n, const int c, const Dtype *x, Dtype* y){
	int c_offset = 0;

		//initialize y
		while(c_offset<c){
			//int idx1 = blockIdx.y * blockDim.x + c_offset + threadIdx.x;
			int idx1 = blockIdx.y * c + c_offset + threadIdx.x;
			if(c_offset + threadIdx.x < c){//WITHOUT THIS: THE C MUST BE MULTIPLE TIMES OF BLOCKDIM.X IN CURRENT IMPLEMENTATION !!!
				y[idx1] = x[idx1]*x[idx1];
			}
			c_offset += blockDim.x;
		}
		__syncthreads();

		//sum along rows
		c_offset=0;
		Dtype res = 0;
		while(c_offset<c){
			int len = (c_offset + blockDim.x)<c ? blockDim.x : (c-c_offset);//valid threads to process
			while(len/2>0){
				if(threadIdx.x<len/2){
					//int idx1 = blockIdx.y * blockDim.x + c_offset + threadIdx.x;
					//int idx2 = blockIdx.y * blockDim.x + c_offset + threadIdx.x + (len+1)/2;
					int idx1 = blockIdx.y * c + c_offset + threadIdx.x;
					int idx2 = blockIdx.y * c + c_offset + threadIdx.x + (len+1)/2;
					y[idx1] += y[idx2];
				}
				__syncthreads();
				len=(len+1)/2;
			}

			//res += y[blockIdx.y * blockDim.x + c_offset];
			res += y[blockIdx.y * c + c_offset];
			c_offset += blockDim.x;
		}
		__syncthreads();

		//copy
		c_offset=0;
		while(c_offset<c){
			//int idx1 = blockIdx.y * blockDim.x + c_offset + threadIdx.x;
			int idx1 = blockIdx.y * c + c_offset + threadIdx.x;
			if(c_offset + threadIdx.x < c){
				if(res){
					y[idx1] = Dtype(sqrt(res));
				}else{
					y[idx1] = Dtype(0);
				}
			}
		  	c_offset += blockDim.x;
		}
}
#define XOFFSET(idx) ((idx)%blk_size_c)
#define YOFFSET(idx) ((idx)/blk_size_c)
//Usage: dim3 block(a,b); dim3 thread(get_threads_per_block,1); col_group_lasso_kernel<<<block,thread,sharemembytesize>>>(n,c,x,y);
//one-D thread block processes two-D data block
template  <typename Dtype>
__global__ void block_group_lasso_kernel(const int n, const int c,
										const Dtype *x, Dtype* y){
	int c_offset = 0;
	const int blk_size_n = n%gridDim.y ? n/gridDim.y+1 : n/gridDim.y;
	const int blk_size_c = c%gridDim.x ? c/gridDim.x+1 : c/gridDim.x;
	//extern __shared__ Dtype shared_mem[];
	//initialize 1D shared memory
	while(c_offset<blk_size_n*blk_size_c){
		int offset_x = XOFFSET(c_offset + threadIdx.x);
		int offset_y = YOFFSET(c_offset + threadIdx.x);
		int x_pos = blockIdx.x * blk_size_c + XOFFSET(c_offset + threadIdx.x);
		int y_pos = blockIdx.y * blk_size_n + YOFFSET(c_offset + threadIdx.x);
		int idx1 = y_pos * c + x_pos;
		//if(x_pos < c && y_pos < n){//WITHOUT THIS: THE C MUST BE MULTIPLE TIMES OF BLOCKDIM.X IN CURRENT IMPLEMENTATION !!!
		if(offset_x < blk_size_c && offset_y < blk_size_n){//WITHOUT THIS: THE C MUST BE MULTIPLE TIMES OF BLOCKDIM.X IN CURRENT IMPLEMENTATION !!!
			y[idx1] = x[idx1]*x[idx1];
			//shared_mem[c_offset + threadIdx.x] = x[idx1]*x[idx1];
		}//else{
			//shared_mem[c_offset + threadIdx.x] = 0;
		//}
		c_offset += blockDim.x;
	}
	__syncthreads();

	//sum along block
	c_offset=0;
	Dtype res = 0;
	while(c_offset<blk_size_n*blk_size_c){
		int len = (c_offset + blockDim.x)<blk_size_n*blk_size_c ? blockDim.x : (blk_size_n*blk_size_c-c_offset);//valid threads to process
		while(len/2>0){
			if(threadIdx.x<len/2){

				int x_pos = blockIdx.x * blk_size_c + XOFFSET(c_offset + threadIdx.x);
				int y_pos = blockIdx.y * blk_size_n + YOFFSET(c_offset + threadIdx.x);
				int idx1 = y_pos * c + x_pos;
				//Dtype sum_elem1 = (x_pos < c && y_pos < n) ? y[idx1] : 0;
				x_pos = blockIdx.x * blk_size_c + XOFFSET(c_offset + threadIdx.x + (len+1)/2);
				y_pos = blockIdx.y * blk_size_n + YOFFSET(c_offset + threadIdx.x + (len+1)/2);
				int idx2 = y_pos * c + x_pos;
				//Dtype sum_elem2 = (x_pos < c && y_pos < n) ? y[idx2] : 0;
				//BUG: we must ALWAYS store this data. Use shared memory with size of blk_size_n*blk_size_c!!!
				y[idx1] += y[idx2];
				//shared_mem[c_offset+threadIdx.x] = sum_elem1+sum_elem2;

				//int idx1 = c_offset + threadIdx.x;
				//int idx2 = c_offset + threadIdx.x + (len+1)/2;
				//shared_mem[idx1] += shared_mem[idx2];
			}
			__syncthreads();
			len=(len+1)/2;
		}

		//res += y[blockIdx.y * c + c_offset];
		int x_pos = blockIdx.x * blk_size_c + XOFFSET(c_offset);
		int y_pos = blockIdx.y * blk_size_n + YOFFSET(c_offset);
		int idx1 = y_pos * c + x_pos;
		res += y[idx1];
		//res += shared_mem[c_offset];
		c_offset += blockDim.x;
	}
	__syncthreads();

	//copy
	c_offset=0;
	while(c_offset<blk_size_n*blk_size_c){
		int offset_x = XOFFSET(c_offset + threadIdx.x);
		int offset_y = YOFFSET(c_offset + threadIdx.x);
		int x_pos = blockIdx.x * blk_size_c + XOFFSET(c_offset + threadIdx.x);
		int y_pos = blockIdx.y * blk_size_n + YOFFSET(c_offset + threadIdx.x);
		int idx1 = y_pos * c + x_pos;
		//if(x_pos < c && y_pos < n){
		if(offset_x < blk_size_c && offset_y < blk_size_n){
			if(res){
				y[idx1] = Dtype(sqrt(res));
			}else{
				y[idx1] = Dtype(0);
			}
		}
	  	c_offset += blockDim.x;
	}
}

template <>
void caffe_gpu_bar_group_lasso<int>(const int n, const int c, const int* x, int* y, bool along_column_or_row){
	NOT_IMPLEMENTED;
}

template <>
void caffe_gpu_bar_group_lasso<unsigned int>(const int n, const int c, const unsigned int* x, unsigned int* y, bool along_column_or_row){
	NOT_IMPLEMENTED;
}

template <>
void caffe_gpu_bar_group_lasso<float>(const int n, const int c, const float* x, float* y, bool along_column_or_row){
	int threads_per_block = Caffe::get_threads_per_block();
	//LOG(INFO)<<"threads_per_block "<<threads_per_block;
	if(along_column_or_row){
		dim3 block(c,1);
		dim3 thread(1,n>threads_per_block ? threads_per_block:n );//CAFFE_CUDA_NUM_THREADS
		col_group_lasso_kernel<<<block,thread>>>(n,c,x,y);
	}else{
		dim3 block(1,n);
		dim3 thread(c>threads_per_block ? threads_per_block:c, 1);//CAFFE_CUDA_NUM_THREADS
		row_group_lasso_kernel<<<block,thread>>>(n,c,x,y);
	}
	CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_bar_group_lasso<double>(const int n, const int c, const double* x, double* y, bool along_column_or_row){
	int threads_per_block = Caffe::get_threads_per_block();
	//LOG(INFO)<<"threads_per_block "<<threads_per_block;
	if(along_column_or_row){
		dim3 block(c,1);
		dim3 thread(1,n>threads_per_block ? threads_per_block:n );//CAFFE_CUDA_NUM_THREADS
		col_group_lasso_kernel<<<block,thread>>>(n,c,x,y);
	}else{
		dim3 block(1,n);
		dim3 thread(c>threads_per_block ? threads_per_block:c, 1);//CAFFE_CUDA_NUM_THREADS
		row_group_lasso_kernel<<<block,thread>>>(n,c,x,y);
	}
	CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_block_group_lasso<float>(const int n, const int c,
		const int blk_size_n, const int blk_size_c,
		const float *x, float* y){
	CHECK_LE(blk_size_n,n);
	CHECK_LE(blk_size_c,c);
	CHECK_EQ(n%blk_size_n,0);
	CHECK_EQ(c%blk_size_c,0);
	int threads_per_block = Caffe::get_threads_per_block();
	//int shared_mem_bytes_per_block = Caffe::get_shared_mem_bytes_per_block();
	const int blk_num_n = (n+blk_size_n-1)/blk_size_n;
	const int blk_num_c = (c+blk_size_c-1)/blk_size_c;
	const int blk_size = blk_size_n*blk_size_c;
	//const int sharedmem_bytes = blk_size*sizeof(float)*2;
	//CHECK_GE(shared_mem_bytes_per_block,sharedmem_bytes);
	dim3 block(blk_num_c,blk_num_n);
	dim3 thread(blk_size>threads_per_block?threads_per_block:blk_size, 1);
	//LOG(INFO)<< "blk_size_n:" << blk_size_n
	//		<< " blk_size_c:" << blk_size_c
	//		<< " blk_num_n:" << blk_num_n
	//		<< " blk_num_c:" << blk_num_c;
	//block_group_lasso_kernel<<<block,thread,sharedmem_bytes>>>(n, c,x,y);
	block_group_lasso_kernel<<<block,thread>>>(n, c,x,y);
	CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_block_group_lasso<double>(const int n, const int c,
		const int blk_size_n, const int blk_size_c,
		const double *x, double* y){
	NOT_IMPLEMENTED;
}
template <>
void caffe_gpu_block_group_lasso<int>(const int n, const int c,
		const int blk_size_n, const int blk_size_c,
		const int *x, int* y){
	NOT_IMPLEMENTED;
}
template <>
void caffe_gpu_block_group_lasso<unsigned int>(const int n, const int c,
		const int blk_size_n, const int blk_size_c,
		const unsigned int *x, unsigned int* y){
	NOT_IMPLEMENTED;
}


template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <typename Dtype>
__global__ void div_checkzero_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = b[index] ? (a[index] / b[index]) : Dtype(0);
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div_checkzero<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_checkzero_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div_checkzero<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
	div_checkzero_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(if_zerout, y[index] = ((x[index] < Dtype(ZEROUT_THRESHOLD) && x[index] > Dtype(-ZEROUT_THRESHOLD) ) ? 1 : 0) );
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(if_nonzerout, y[index] = ((x[index] >= Dtype(ZEROUT_THRESHOLD) || x[index] <= Dtype(-ZEROUT_THRESHOLD) ) ? 1 : 0) )
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(eltwise_multi, y[index] = y[index]*x[index] )
void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
